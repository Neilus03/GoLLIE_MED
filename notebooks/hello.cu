
#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void helloFromGPU()
{
    printf("Hello World from GPU!\n");
}

int main()
{
    printf("Hello World from CPU!\n");
    helloFromGPU<<<1, 10>>>();
    hipDeviceSynchronize();
    return 0;
}
